#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <opencv4/opencv2/opencv.hpp>
#include <opencv4/opencv2/core.hpp>
#include <opencv4/opencv2/imgcodecs.hpp>


__global__ void blurKernel(unsigned char *d_input, unsigned char *d_output, int width, int height) {
    int idx = threadIdx.x+blockDim.x*blockIdx.x;
    int idy = threadIdx.y+blockDim.y*blockIdx.y;
    if (idx < width && idy < height) {
        int greyScaleOffset = idy * width + idx;
        int rgbOffset = greyScaleOffset * 3;

        unsigned char r = d_input[rgbOffset];
        unsigned char g = d_input[rgbOffset + 1];
        unsigned char b = d_input[rgbOffset + 2];

        d_output[greyScaleOffset] = 0.21f * r + 0.72f * g + 0.07f * b;
        
    }
}

int main(){
    cv::Mat input = cv::imread("test.jpg");
    
    if (input.empty()) {
        printf("Error: Could not read input image\n");
        return -1;
    }

    int width = input.cols;
    int height = input.rows;
    
    // Allocate host memory for output
    cv::Mat output(height, width, CV_8UC1);

    // Allocate device memory
    unsigned char *d_input, *d_output;
    hipMalloc(&d_input, width * height * 3 * sizeof(unsigned char));  // RGB input
    hipMalloc(&d_output, width * height * sizeof(unsigned char));     // Grayscale output
    cudaCheckError();

    // Copy input image to device
    hipMemcpy(d_input, input.data, width * height * 3 * sizeof(unsigned char), 
               hipMemcpyHostToDevice);
    cudaCheckError();

    // Set up grid and block dimensions
    dim3 blockDim(16, 16);  // 256 threads per block
    dim3 gridDim((width + blockDim.x - 1) / blockDim.x, 
                 (height + blockDim.y - 1) / blockDim.y);

    // Launch kernel
    blurKernel<<<gridDim, blockDim>>>(d_input, d_output, width, height);
    cudaCheckError();

    // Copy result back to host
    hipMemcpy(output.data, d_output, width * height * sizeof(unsigned char), 
               hipMemcpyDeviceToHost);
    cudaCheckError();

    // Save output image
    cv::imwrite("output.jpg", output);

    // Clean up
    hipFree(d_input);
    hipFree(d_output);
    cudaCheckError();

    return 0;
}


