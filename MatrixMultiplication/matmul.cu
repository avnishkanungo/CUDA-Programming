#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void matMul(float* a, float* b, float* c, int a_rows, int a_cols, int b_cols){     //dont need to define b_rows as that will need to be same as a_cols for matrix mul to works
    int rows = threadIdx.y + blockDim.y * blockIdx.y;
    int cols = threadIdx.x + blockDim.x * blockIdx.x;

    if(rows<a_rows && cols<b_cols){
        float sum = 0.0f;
        for(int k=0;k<a_cols;k++){
            sum += a[rows*a_cols+k]*b[k*b_cols+cols];
        }
        c[rows*b_cols+cols] = sum;
    }

}


void initializeMatrix(float* d_a, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            d_a[i*cols+j] = rand()%10;
        }
    }
}

void printMatrix(float* d_a, int rows, int cols){
    for(int i=0; i<rows; i++){
        for(int j=0; j<cols; j++){
            printf("%f ", d_a[i*cols+j]);
        }
        printf("\n");
    }
    printf("\n");
}

int main(){
    float *d_a, *d_b, *d_c;
    float *h_a, *h_b, *h_c;

    int a_rows,a_cols,b_rows,b_cols;

    printf("Enter the value for the rows and colums for d_a:");
    scanf("%d %d", &a_rows,&a_cols);

    printf("Enter the value for the rows and colums for d_b:");
    scanf("%d %d", &b_rows,&b_cols);

    if (a_cols != b_rows){
        printf("Error");
        return -1;
    }

    h_a = (float*)malloc(a_rows*a_cols*sizeof(float));
    h_b = (float*)malloc(b_rows*b_cols*sizeof(float));
    h_c = (float*)malloc(a_rows*b_cols*sizeof(float));

    initializeMatrix(h_a,a_rows,a_cols);
    initializeMatrix(h_b,b_rows,b_cols);

    hipMalloc((void**)&d_a,a_rows*a_cols*sizeof(float));
    hipMalloc((void**)&d_b,b_rows*b_cols*sizeof(float));
    hipMalloc((void**)&d_c,a_rows*b_cols*sizeof(float));

    hipMemcpy(d_a,h_a,a_rows*a_cols*sizeof(float),hipMemcpyHostToDevice);
    hipMemcpy(d_b,h_b,b_rows*b_cols*sizeof(float),hipMemcpyHostToDevice);

    dim3 threadsInBlock(16,16);
    dim3 blocksInGrid((a_rows+threadsInBlock.x-1)/threadsInBlock.x,
                      (b_cols+threadsInBlock.y-1)/threadsInBlock.y);

    matMul<<<blocksInGrid,threadsInBlock>>>(d_a,d_b,d_c,a_rows,a_cols,b_cols);

    hipMemcpy(h_c,d_c,a_rows*b_cols*sizeof(float),hipMemcpyDeviceToHost);
    printMatrix(h_c,a_rows,b_cols);
}