#include "hip/hip_runtime.h"

M/teamspace/studios/this_studio/MatrixAdditionAndAssignment/matrix_addition.cu�#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void matrixAdd(int* a, int* b, int* c, int w, int l) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    if (idx<w && idy<l){
        c[idy*w+idx] = a[idy*w+idx] + b[idy*w+idx];
    };
}

int main() {
    const int w = 64;
    const int l = 64;
    size_t size = w * l * sizeof(int);

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    for (int i = 0; i < w * l; i++) {
        h_a[i] = i;  // Random values for array a
        h_b[i] = i*2;  // Random values for array b
    }

    hipMalloc((void**)&d_a,size);
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_b,size);
    hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_c,size);

    dim3 threadsInBlock(16,16);
    dim3 blocksInGrid(
        (w + threadsInBlock.x-1)/threadsInBlock.x,
        (l + threadsInBlock.x-1)/threadsInBlock.y
    );

    matrixAdd<<<blocksInGrid,threadsInBlock>>>(d_a,d_b,d_c,w,l);

    hipDeviceSynchronize();

    hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    for(int i=0;i<10;i++){
        for(int j=0; j<10; j++){
            printf("%d+%d=%d\n",h_a[i*l+j],h_b[i*l+j],h_c[i*l+j]);
        }
        
    };

    return 0;

}0*$0ecbdd1c-c002-4afa-90d5-c50c51adf33c082 *$8d40c9ad-671b-4f52-9948-4654d897549508'2'* *$8d40c9ad-671b-4f52-9948-4654d897549508*R 2RV *$171088a0-cc15-4180-8d15-fa69cfd5ed3f08VY2Yb *$171088a0-cc15-4180-8d15-fa69cfd5ed3f08bc2cj *$171088a0-cc15-4180-8d15-fa69cfd5ed3f08jk kr 	r� 2��*$0b2039e8-6e26-4620-bfbb-a9439effecb408
�� 
�� 
�� 
�� 
�� 
�� 
�� 2��*$9be76487-46cb-48a4-aca5-3c43e9bfb5cf08
�� 
�� ��
�� ��
�� 
�� 
�� ��
�� ��
�� 
�� 
�� ��
�� ��
�� ��
�� 
�� 
�� ��
�� ��
�� ��
�� ��
�� ��
�� ��
�� 
�� 
�� 
�� 
�� ��
�� 
�� ��
�� 
�� 
�� 
�� 
�� ��	
�	�	 �	�	
�	�	 
�	�	 �	�	
�	�	 �	�	
�	�	 
�	�	 
�	�	 �	�	
�	�	 �	�	
�	�	 
�	�
 
�
�
 
�
�
 
�
� 
�� 
�� ��
�� ��
�� 
�� 2Tfile:///teamspace/studios/this_studio/MatrixAdditionAndAssignment/matrix_addition.cu