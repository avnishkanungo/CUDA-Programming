#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void matrixAdd(int* a, int* b, int* c, int w, int l) {
    int idx = blockIdx.x*blockDim.x + threadIdx.x;
    int idy = blockIdx.y*blockDim.y + threadIdx.y;
    if (idx<w && idy<l){
        c[idy*w+idx] = a[idy*w+idx] + b[idy*w+idx];
    };
}

int main() {
    const int w = 64;
    const int l = 64;
    size_t size = w * l * sizeof(int);

    int *h_a, *h_b, *h_c;
    int *d_a, *d_b, *d_c;

    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);

    for (int i = 0; i < w * l; i++) {
        h_a[i] = i;  // Random values for array a
        h_b[i] = i*2;  // Random values for array b
    }

    hipMalloc((void**)&d_a,size);
    hipMemcpy(d_a,h_a,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_b,size);
    hipMemcpy(d_b,h_b,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&d_c,size);

    dim3 threadsInBlock(16,16);
    dim3 blocksInGrid(
        (w + threadsInBlock.x-1)/threadsInBlock.x,
        (l + threadsInBlock.x-1)/threadsInBlock.y
    );

    matrixAdd<<<blocksInGrid,threadsInBlock>>>(d_a,d_b,d_c,w,l);

    hipDeviceSynchronize();

    hipMemcpy(h_c,d_c,size,hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    free(h_a);
    free(h_b);
    free(h_c);

    for(int i=0;i<10;i++){
        for(int j=0; j<10; j++){
            printf("%d+%d=%d\n",h_a[i*l+j],h_b[i*l+j],h_c[i*l+j]);
        }
        
    };

    return 0;

}