#include <stdio.h>
#include <hip/hip_runtime.h>

// CUDA kernel for vector addition
__global__ void vectorAdd(int *a, int *b, int *c, int n) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < n) {
        c[idx] = a[idx] + b[idx];
    }
}

int main() {
    const int N = 1000;
    size_t size = N * sizeof(int);
    
    // Host arrays
    int *h_a, *h_b, *h_c;
    // Device arrays
    int *d_a, *d_b, *d_c;
    
    // Allocate host memory
    h_a = (int*)malloc(size);
    h_b = (int*)malloc(size);
    h_c = (int*)malloc(size);
    
    // Initialize arrays
    for (int i = 0; i < N; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }
    
    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);
    
    // Copy data to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
    // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);
    
    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // Verify results
    for (int i = 0; i < 5; i++) {
        printf("%d + %d = %d\n", h_a[i], h_b[i], h_c[i]);
    }
    
    // Free memory
    free(h_a);
    free(h_b);
    free(h_c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    
    return 0;
}